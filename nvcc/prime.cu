
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void kernel( int* result )
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int number = result[i];
	for(int p = 2; p <= number/2; p++)
	{
		if(number % p == 0) return;
	}
	result[i] = -number;	// is prime
}

double diff_sec(time_t start, time_t end){ return (double)(end - start)/CLOCKS_PER_SEC; }

int main(int argc, char *argv[])
{
	int limit = 10000;
	if(argc == 2) limit = atoi(argv[1]);
	const int STEP = 1024*16;

	clock_t start = clock();
	int *d_buffer;
	int *buffer;
	hipMalloc((void**)&d_buffer, STEP*sizeof(int));
	buffer = new int[STEP];
	int count = 0;
	for(int number = 2; ; number += STEP)
	{
		clock_t time0 = clock();

		for(int i = 0; i < STEP; i++) buffer[i] = number + i;
		hipMemcpy(d_buffer, buffer, STEP*sizeof(int), hipMemcpyHostToDevice);
		clock_t time1 = clock();

		dim3 block(512, 1, 1);
		dim3 grid(STEP/block.x, 1, 1);
		kernel<<<grid,block>>>(d_buffer);
		clock_t time2 = clock();

		hipMemcpy(buffer, d_buffer, STEP*sizeof(int), hipMemcpyDeviceToHost);
		clock_t time3 = clock();

		for(int i = 0; i < STEP; i++)
		{
			if(buffer[i] > 0) continue;
			count++;
//printf("\t%d", buffer[i]);
			if(count == limit)
			{
				printf("%dth prime number is %d\n", limit, -buffer[i]);
				break;
			}
		}
		printf("1=%.4f,2=%.4f,3=%.4f\n", diff_sec(time0,time1), diff_sec(time1,time2), diff_sec(time2,time3));
		if(count == limit) break;
	}

	hipFree(d_buffer);
	clock_t end = clock();

	printf("[nvcc] duration is %.2f sec\n", diff_sec(start,end));
	return 0;
}

