
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__ void kernel( int* result )
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	result[i] = i;
	for(int p = 2; p <= i/2; p++)
	{
		if(i % p == 0){ result[i] = 0; break; }
	}
}

double diff_sec(time_t start, time_t end)
{
	return (double)(end - start)/CLOCKS_PER_SEC;
}

int main( void )
{
	clock_t start = clock();
	const int MAX = 1024*1024;
	int *d_buffer;
	int *buffer;
	hipMalloc((void**)&d_buffer, MAX*sizeof(int));
	buffer = new int[MAX];
	clock_t time1 = clock();

	dim3 block(512, 1, 1);
	dim3 grid(MAX/block.x, 1, 1);

	kernel<<<grid,block>>>(d_buffer);
	clock_t time2 = clock();

	hipMemcpy(buffer, d_buffer, MAX*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_buffer);
	clock_t end = clock();
	int count = 1;
	int target = 0;
	for(int i = 0; i < MAX; i++)
	{
		if(buffer[i]==0) continue;
//		printf("%d=%d\n", count, buffer[i]);
		if(count == 10000) target = buffer[i];
		count++;
	}
	printf("total=%.2f,warm-up=%.2f,gpu=%.2f,cool-down=%.2f\n", diff_sec(start,end), diff_sec(start,time1), diff_sec(time1,time2), diff_sec(time2,end));
	printf("10000th prime number is %d\n[nvcc] duration is %.2f sec\n", target, diff_sec(start,end) * 10000.0/MAX);
	return 0;
}

