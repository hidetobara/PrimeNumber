
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__ void kernel( int* result )
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	result[i] = i;
	for(int p = 2; p <= i/2; p++)
	{
		if(i % p == 0){ result[i] = 0; break; }
	}
}

int main( void )
{
	clock_t start = clock();
	const int MAX = 1024*1024;
	int *d_buffer;
	int *buffer;
	hipMalloc((void**)&d_buffer, MAX*sizeof(int));
	buffer = new int[MAX];

	dim3 block(512, 1, 1);
	dim3 grid(MAX/block.x, 1, 1);

	kernel<<<grid,block>>>(d_buffer);
	hipMemcpy(buffer, d_buffer, MAX*sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_buffer);
	clock_t end = clock();
	int count = 1;
	int target = 0;
	for(int i = 0; i < MAX; i++)
	{
		if(buffer[i]==0) continue;
//		printf("%d=%d\n", count, buffer[i]);
		if(count == 10000) target = buffer[i];
		count++;
	}
	printf("total=%.2f\n", (double)(end-start)/CLOCKS_PER_SEC);
	printf("10000th prime number is %d : %.2f sec\n", target, (double)(end-start)/CLOCKS_PER_SEC * 10000.0/count);
	return 0;
}

